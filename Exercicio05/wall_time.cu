#ifdef _WIN32
#include <time.h>
#else

#include <hip/hip_runtime.h>
#include <sys/time.h>
#endif

double wall_time(void)
{
#ifdef _WIN32
  return (double)((double)clock() / (double)CLOCKS_PER_SEC);
#else
  struct timeval tv;
  struct timezone tz;
  gettimeofday(&tv, &tz);
  return(tv.tv_sec + tv.tv_usec/1000000.0);
#endif
}