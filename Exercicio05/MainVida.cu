
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// interfaces

double wall_time(void);
void UmaVida(int*, int*, int);
__global__ void UmaVidaGpu(int*, int*, int);
void InitTabul(int*, int*, int);
void DumpTabul(int*, int, int, int, char*);
int Correto(int*, int);



// Trafega um veleiro de tamanho tam ao longo do tabuleiro
// Tamanho tam eh produto dos argumentos de entrada tamBlk e nBlk
// Tamanho minimo eh MinTam




int main(int argc, char *argv[]) {
#define MinTam 4
  int i;
  int tam, tamBlk, nBlk;
  int* tabulIn;
  int* tabulOut;
  size_t size;
  double t0, t1, t2, t3;
  char msg[16];
  
  // obtem tamanho do tabuleiro

  if (argc != 3) {
    printf(" uso: <exec> <celulas por bloco> <quantos blocos>\n");
    exit(-1);
  }
  tamBlk = atoi(argv[1]);
  nBlk = atoi(argv[2]);
  tam = nBlk*tamBlk;
  size = (tam+2)*(tam+2)*sizeof(int);

  // tamanho minimo
  if (tam < MinTam) {
    printf("**ERRO** tamanho %d menor que o minimo %d\n", tam, MinTam);
    exit(-1);
  }

  // CPU
  // aloca e inicializa tabuleiros

  t0 = wall_time();
  tabulIn  = (int *) malloc (size);
  tabulOut = (int *) malloc (size);
  InitTabul(tabulIn, tabulOut, tam);

  // dump tabuleiro inicial

  sprintf(msg,"Inicial CPU");
  DumpTabul(tabulIn, tam, 1, 4, msg);

  // avanca geracoes

  t1 = wall_time();
  for (i=0; i<2*(tam-3); i++) {
    UmaVida (tabulIn, tabulOut, tam);
    UmaVida (tabulOut, tabulIn, tam);
  }
  t2 = wall_time();

  // dump tabuleiro final

  sprintf(msg,"Final CPU");
  DumpTabul(tabulIn, tam, tam-3, tam, msg);

  // Correcao na CPU

  if (Correto(tabulIn, tam)) 
    printf("**RESULTADO CORRETO NA CPU**\n");
  else
    printf("**RESULTADO ERRADO NA CPU**\n");

  t3 = wall_time();

  // Tempos na CPU

  printf("tam=%d; tempos na CPU: init=%f, comp=%f, fim=%f, tot=%f \n", 
  //js printf("module.exports = {tam:%d, \n")
  //js printf("cpu: {init:%f, comp:%f, fim:%f, tot:%f }, \n", 
	 tam, t1-t0, t2-t1, t3-t2, t3-t0);

  // GPU
  // aloca e inicializa tabuleiros
  int* tabulIn_d;
  int* tabulOut_d;

  t0 = wall_time();
  tabulIn  = (int *) malloc (size);
  hipMemcpy(tabulIn_d, tabulIn, size*sizeof(int), hipMemcpyHostToDevice);
  tabulOut = (int *) malloc (size);
  hipMemcpy(tabulOut_d, tabulOut, size*sizeof(int), hipMemcpyHostToDevice);
  dim3 dB (tamBlk, tamBlk);
  dim3 dG (nBlk, nBlk);
  InitTabul(tabulIn, tabulOut, tam);

  // dump tabuleiro inicial

  sprintf(msg,"Inicial GPU");
  DumpTabul(tabulIn, tam, 1, 4, msg);

  // avanca geracoes

  t1 = wall_time();
  for (i=0; i<2*(tam-3); i++) {
    UmaVidaGpu <<< dG, dB >>> (tabulIn, tabulOut, tam);
    UmaVidaGpu <<< dG, dB >>> (tabulOut, tabulIn, tam);
  }
  t2 = wall_time();

  // Pega as variaveis de volta da gpu
  hipMemcpy(tabulIn, tabulIn_d, size*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(tabulOut, tabulOut_d, size*sizeof(int), hipMemcpyDeviceToHost);

  // dump tabuleiro final

  sprintf(msg,"Final GPU");
  DumpTabul(tabulIn, tam, tam-3, tam, msg);

  // Correcao na GPU

  if (Correto(tabulIn, tam)) 
    printf("**RESULTADO CORRETO NA GPU**\n");
  else
    printf("**RESULTADO ERRADO NA GPU**\n");

  t3 = wall_time();

  // Tempos na GPU

  printf("tam=%d; tempos na GPU: init=%f, comp=%f, fim=%f, tot=%f \n", 
  //js printf("gpu: {init:%f, comp:%f, fim:%f, tot:%f },\n", 
	 tam, t1-t0, t2-t1, t3-t2, t3-t0);

  //js printf("}\n")

  free(tabulIn);
  free(tabulOut);

  exit(0);    
}
